#include "hip/hip_runtime.h"
#include "timestamps.cuh"
#include "common.cuh"

#include <stdexcept>
#include <iostream>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void* cuda_malloc(size_t size) {
  void* devPtr;
  CUDA_CHECK(hipMalloc(&devPtr, size));
  return devPtr;
}

static const int NT = 128;
static const int VT = 4;

// static const int NT = 512;
// static const int VT = 1;

static const int BIG_ARG_SIZE = 18 * 3;

struct BigArg {
  int big_arg[BIG_ARG_SIZE];
};

__launch_bounds__(NT, 4)
__global__
void add_kernel(float* __restrict__ out, const float* __restrict__ x, const float* __restrict__ y, int N, BigArg big) {
  int tid = threadIdx.x;
  int cta = blockIdx.x;
  int nv = NT * VT;
  int start = nv * cta;
  int end = min(N, nv * (cta + 1));
  int count = end - start;
  for (int i = 0; i < BIG_ARG_SIZE; i++) {
    start += big.big_arg[i];
  }
  if (count >= NT * VT) {
    int linearIndex = start + tid;
    #pragma unroll
    for (int i = 0; i < VT; i++) {
      out[linearIndex] = x[linearIndex] + y[linearIndex];
      linearIndex += NT;
    }
  } else {
    // assert(0);
  }
}

static void verify(float* out_cuda, float* x_cuda, float* y_cuda, int N) {
  float* x = (float*)malloc(N * sizeof(float));
  float* y = (float*)malloc(N * sizeof(float));
  float* out = (float*)malloc(N * sizeof(float));
  CUDA_CHECK(hipMemcpy(x, x_cuda, N * sizeof(float), hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(y, y_cuda, N * sizeof(float), hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(out, out_cuda, N * sizeof(float), hipMemcpyDefault));
  bool non_zero = false;
  for (int i = 0; i < N; i++) {
    if (out[i] != x[i] + y[i]) {
      throw std::runtime_error(std::string("error at ") + std::to_string(i));
    }
    if (x[i] != 0 && y[i] != 0) {
      non_zero = true;
    }
  }
  if (!non_zero) {
    throw std::runtime_error("all zero");
  }
  std::cout << "OK\n";
}

static uint64_t x = 7; /* The state can be seeded with any value. */

uint64_t next() {
	uint64_t z = (x += 0x9e3779b97f4a7c15);
	z = (z ^ (z >> 30)) * 0xbf58476d1ce4e5b9;
	z = (z ^ (z >> 27)) * 0x94d049bb133111eb;
	return z ^ (z >> 31);
}

float next_float() {
  return (float)((next() >> 11) * (1. / (UINT64_C(1) << 53)));
}

static void fill_random(float* out_cuda, int N) {
  float* cpu = (float*)malloc(N * sizeof(float));
  for (int i = 0; i < N; i++) {
    cpu[i] = next_float();
  }
  CUDA_CHECK(hipMemcpy(out_cuda, cpu, N * sizeof(float), hipMemcpyDefault));
  free(cpu);
}

int main(int argc, char* argv[]) {
  static const int N = 1024 * 1024 * 10;
  auto x = (float*)cuda_malloc(N * sizeof(float));
  auto y = (float*)cuda_malloc(N * sizeof(float));
  auto res = (float*)cuda_malloc(N * sizeof(float));

  fill_random(x, N);
  fill_random(y, N);

  hipDeviceProp_t deviceProperties;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProperties, 0));

  std::cout << "multiProcessorCount: " << deviceProperties.multiProcessorCount << "\n";

  dim3 block(NT);
  dim3 grid(N / block.x / VT);

  BigArg big;
  for (int i = 0; i < BIG_ARG_SIZE; i++) {
    big.big_arg[i] = next_float() > 100 ? next_float() : 0;
  }

  CUDA_CHECK(hipDeviceSynchronize());
  for (int i = 0; i < 10; i++) {
    cuda_timestamp start;
    add_kernel<<<grid, block>>>(res, x, y, N, big);
    std::cout << "time " << start.elapsed_time() << "\n";
  }

  verify(res, x, y, N);

  // cuda_timestamp end;
  // int64_t ts[10];
  // for (int i = 0; i < 10; i++) {
  //   timestamp ss;
  //   // start.record();
  //   // end.record();
  //   ts[i] = ss.elapsed_time();
  // }
  // for (int i = 0; i < 10; i++) {
  //   std::cout << ts[i] << "\n";
  // }

  return 0;
}
