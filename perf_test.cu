#include "hip/hip_runtime.h"
#include "timestamps.cuh"
#include "common.cuh"
#include "offset_info.cuh"

#include <stdexcept>
#include <iostream>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void* cuda_malloc(size_t size) {
  void* devPtr;
  CUDA_CHECK(hipMalloc(&devPtr, size));
  return devPtr;
}

static const int NT = 256;
static const int VT = 4;

// static const int NT = 512;
// static const int VT = 1;


__launch_bounds__(NT, 4)
__global__
void add_kernel(float* __restrict__ out, const float* __restrict__ x, const float* __restrict__ y, int N,
                OffsetInfo o1, StrideInfo s1, StrideInfo s2, StrideInfo s3) {
  int tid = threadIdx.x;
  int cta = blockIdx.x;
  int nv = NT * VT;
  int start = nv * cta;
  int end = min(N, nv * (cta + 1));
  int count = end - start;
  if (count >= NT * VT) {
    int linearIndex = start + tid;
    #pragma unroll
    for (int i = 0; i < VT; i++) {
      int idx1, idx2, idx3;
      o1.get(linearIndex, s1, s2, s3, &idx1, &idx2, &idx3);

      out[idx1] = x[idx2] + y[idx3];
      linearIndex += NT;
    }
  } else {
    // assert(0);
  }
}

static void verify(float* out_cuda, float* x_cuda, float* y_cuda, int N) {
  float* x = (float*)malloc(N * sizeof(float));
  float* y = (float*)malloc(N * sizeof(float));
  float* out = (float*)malloc(N * sizeof(float));
  CUDA_CHECK(hipMemcpy(x, x_cuda, N * sizeof(float), hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(y, y_cuda, N * sizeof(float), hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(out, out_cuda, N * sizeof(float), hipMemcpyDefault));
  bool non_zero = false;
  for (int i = 0; i < N; i++) {
    if (out[i] != x[i] + y[i]) {
      throw std::runtime_error(std::string("error at ") + std::to_string(i));
    }
    if (x[i] != 0 && y[i] != 0) {
      non_zero = true;
    }
  }
  if (!non_zero) {
    throw std::runtime_error("all zero");
  }
  std::cout << "OK\n";
}

static uint64_t x = 7; /* The state can be seeded with any value. */

uint64_t next() {
	uint64_t z = (x += 0x9e3779b97f4a7c15);
	z = (z ^ (z >> 30)) * 0xbf58476d1ce4e5b9;
	z = (z ^ (z >> 27)) * 0x94d049bb133111eb;
	return z ^ (z >> 31);
}

float next_float() {
  return (float)((next() >> 11) * (1. / (UINT64_C(1) << 53)));
}

static void fill_random(float* out_cuda, int N) {
  float* cpu = (float*)malloc(N * sizeof(float));
  for (int i = 0; i < N; i++) {
    cpu[i] = next_float();
  }
  CUDA_CHECK(hipMemcpy(out_cuda, cpu, N * sizeof(float), hipMemcpyDefault));
  free(cpu);
}

int main(int argc, char* argv[]) {
  static const int N = 1024 * 1024 * 10;
  int64_t sizes[] = {10, 32, 32, 32, 32};
  int64_t strides[] = {1, 10, 320, 10240, 327680};

  auto offset = OffsetInfo(5, sizes);
  auto stride_info = StrideInfo(5, strides);

  auto x = (float*)cuda_malloc(N * 2 * sizeof(float));
  auto y = (float*)cuda_malloc(N * 2 * sizeof(float));
  auto res = (float*)cuda_malloc(N * 2 * sizeof(float));

  fill_random(x, N);
  fill_random(y, N);

  hipDeviceProp_t deviceProperties;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProperties, 0));

  std::cout << "multiProcessorCount: " << deviceProperties.multiProcessorCount << "\n";

  dim3 block(NT);
  dim3 grid(N / block.x / VT);

  CUDA_CHECK(hipDeviceSynchronize());
  for (int i = 0; i < 10; i++) {
    cuda_timestamp start;
    add_kernel<<<grid, block>>>(res, x, y, N, offset, stride_info, stride_info, stride_info);
    std::cout << "time " << start.elapsed_time() << "\n";
  }

  verify(res, x, y, N);

  // cuda_timestamp end;
  // int64_t ts[10];
  // for (int i = 0; i < 10; i++) {
  //   timestamp ss;
  //   // start.record();
  //   // end.record();
  //   ts[i] = ss.elapsed_time();
  // }
  // for (int i = 0; i < 10; i++) {
  //   std::cout << ts[i] << "\n";
  // }

  return 0;
}
